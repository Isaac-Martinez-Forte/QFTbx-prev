#include "hip/hip_runtime.h"
//include C++
#include <iostream>
#include <vector>
#include <complex>
#include <string>
#include <cmath>


//include CUDA
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>


#define inf 0x7f800000
#define pi 3.1416

typedef float2 Complex;

using namespace std;
using namespace thrust;


int threadsPerBlock = 128;
size_t size_double = 0;
int blocksPerGrid = 0;

//Función principal para resolver e_hull con cuda.
extern "C"
vector<complex<double> > e_hull_cuda(vector <complex <double> > puntos, float epsilon);

//Función que encuentra el punto segundo de la nube de puntos.
static __global__ void buscar_segundo_kernel(Complex primero, Complex * puntos, float epsilon,
                                             double * retorno, int n_puntos);
//Función que encuentra el punto siguiente de la nube de puntos.
static __global__ void buscar_siguiente_kernel(Complex primero, Complex segundo, Complex * puntos, float epsilon,
                                               double * retorno, int n_puntos);


//Funciones internas
//int buscar_primero(host_vector<double> vector_buscar_primero_h);
int buscar_segundo (Complex * puntos_d, Complex complejo_primero, float epsilon, int numElements);
int buscar_siguiente (Complex complejo_primero, Complex complejo_segundo, Complex * puntos_d, float epsilon, int numElements,
                      double * retorno_siguiente_d);

//Funciones internas GPU
static __device__ __host__ inline double abs(Complex a);
static __device__ __host__ inline double arg(Complex a);
static __device__ __host__ inline Complex operator-(Complex a, Complex b);
static __device__ __host__ inline double arg(Complex a);
static __device__ __host__ inline Complex operator/(Complex a, Complex b);
static __device__ __host__ inline bool operator!=(Complex a, Complex b);


vector <complex <double> > e_hull_cuda(vector<complex<double> > puntos, float epsilon){

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    vector <int> indice_contorno;

    // Creamos los tamaños de los distintos vectores
    int numElements = puntos.size();
    size_t size_complex = numElements * sizeof(Complex);
    size_double = numElements * sizeof(double);
    blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    ///////////////////////////////////////////////////////////////////////////////
    ////////////////////////////// Buscar Primero /////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////

    //Creamos el vector de complejos
    Complex * puntos_h = (Complex *) malloc(size_complex);

    //Creamos el vector del host para buscar el primer número
    host_vector<double> vector_buscar_primero_h (numElements);
    
    double numDe = -inf;
    int posicion_primero = 0;

    for (int i = 0; i < numElements; i++){
            
        puntos_h[i].x = real(puntos.at(i));
	
        puntos_h[i].y = imag(puntos.at(i));
	
	if (puntos_h[i].y > numDe){
            posicion_primero = i;
            numDe = puntos_h[i].y;
        }

        //vector_buscar_primero_h[i] =  puntos.at(i).real();
    }

    //int posicion_primero = buscar_primero(vector_buscar_primero_h);

    indice_contorno.push_back(posicion_primero);

    //Guardamos el primero número complejo del contorno
    Complex complejo_primero;
    complejo_primero.x = puntos_h[posicion_primero].x;
    complejo_primero.y = puntos_h[posicion_primero].y;

    ///////////////////////////////////////////////////////////////////////////////
    ////////////////////////////// Buscar Segundo /////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////

    Complex * puntos_d = NULL;
    err = hipMalloc((void **)&puntos_d, size_complex);

    if (err != hipSuccess)
    {
        cerr << "Error al reservar memoria en CUDA (error code " <<  hipGetErrorString(err) << ")!" << endl ;
        exit(EXIT_FAILURE);
    }

    //Copiamos a la memoria de CUDA los datos.
    hipMemcpy(puntos_d, puntos_h, size_complex, hipMemcpyHostToDevice);

    int posicion_segundo = buscar_segundo(puntos_d, complejo_primero, epsilon, numElements);

    if (posicion_segundo < 0){
        hipFree(puntos_d);
        free(puntos_h);

        vector <complex <double> > vector_nulo;
        return vector_nulo;
    }

    indice_contorno.push_back(posicion_segundo);

    //Preparamos el complejo segundo.
    Complex complejo_segundo;
    complejo_segundo.x = puntos_h[posicion_segundo].x;
    complejo_segundo.y = puntos_h[posicion_segundo].y;


    ///////////////////////////////////////////////////////////////////////////////
    ///////////////////////////// Buscar Siguiente ////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////


    int punto_previo = posicion_primero;
    int punto_actual = posicion_segundo;


    //Creamos las estructuras necesarias para guardar el retorno del kernel
    double * retorno_siguiente_d = NULL;
    hipMalloc((void **)&retorno_siguiente_d, size_double);

    int punto_siguiente = buscar_siguiente(complejo_primero, complejo_segundo, puntos_d,
                                           epsilon, numElements, retorno_siguiente_d);

    if (punto_siguiente < 0){
        hipFree(retorno_siguiente_d);
        hipFree(puntos_d);
        free(puntos_h);

        vector <complex <double> > vector_nulo;
        return vector_nulo;
    }

    int contador = 2;

    int MAXP = numElements;

    while (posicion_primero != punto_actual || posicion_segundo != punto_siguiente){

        indice_contorno.push_back(punto_siguiente);
        contador++;

        if (contador > MAXP){
            break;
        }

        punto_previo = punto_actual;
        punto_actual = punto_siguiente;

        ////////////////////////////////////////////////////////////////////////////
        //Preparamos los números complejos anteriores
        Complex complejo_previo;
        complejo_previo.x = puntos_h[punto_previo].x;
        complejo_previo.y = puntos_h[punto_previo].y;


        Complex complejo_actual;
        complejo_actual.x = puntos_h[punto_actual].x;
        complejo_actual.y = puntos_h[punto_actual].y;

        punto_siguiente = buscar_siguiente(complejo_previo, complejo_actual, puntos_d,
                                           epsilon, numElements, retorno_siguiente_d);


        if (punto_siguiente < 0){
            hipFree(retorno_siguiente_d);
            hipFree(puntos_d);
            free(puntos_h);

            vector <complex <double> > vector_nulo;
            return vector_nulo;
        }

        ////////////////////////////////////////////////////////////////////////////

    }

    ///////////////////////////////////////////////////////////////////////////////


    hipFree(retorno_siguiente_d);
    hipFree(puntos_d);
    free(puntos_h);

    /*device_vector <int> indice_contorno_d = indice_contorno;

    device_vector<int>::iterator final = thrust::unique(thrust::host, indice_contorno_d.begin(), indice_contorno_d.end());
    vector <complex <double> > puntos_contorno;*/



    vector <int>  aux;
    bool isRepetido = false;

    for (int i = 0; i < indice_contorno.size(); i++) {
      
        isRepetido = false;
      
        for (int j = 0;j < aux.size(); j++) {
            if (indice_contorno.at(i) == indice_contorno.at(j)){
                isRepetido = true;
                break;
            }
        }
        if (!isRepetido){
            aux.push_back(indice_contorno.at(i));
        }
    }

    vector <complex <double> > puntos_contorno;

    for (int i = 0; i < aux.size(); i++){
        puntos_contorno.push_back(puntos.at(aux.at(i)));
    }

    return puntos_contorno;
}

/*int buscar_primero(host_vector<double> vector_buscar_primero_h){
    //copiamos los valores al vector del device para buscar el primer número
    device_vector<double> vector_buscar_primero_d = vector_buscar_primero_h;

    //Buscamos el máximo.
    device_vector<double>::iterator iter = max_element(vector_buscar_primero_d.begin(), vector_buscar_primero_d.end());

    //Recuperamos la posición del complejo primero.
    int posicion_primero = iter - vector_buscar_primero_d.begin();

    return posicion_primero;
}*/

int buscar_segundo (Complex * puntos_d, Complex complejo_primero, float epsilon, int numElements){

    //Asignamos memoria el vector de fases de CUDA
    double * retorno_segundo_d = NULL;
    hipMalloc((void **) &retorno_segundo_d, size_double);

    // Lanzamos la ejecución del kernel

    buscar_segundo_kernel <<<blocksPerGrid, threadsPerBlock>>> (complejo_primero, puntos_d, epsilon,
                                                                retorno_segundo_d, numElements);
    hipGetLastError();

    device_ptr<double> d_vec (retorno_segundo_d);
    device_ptr<double> d_vec_final = d_vec + numElements;
    device_ptr<double> result = min_element(d_vec, d_vec_final);

    //Recuperamos la posición del valor mínimo.
    unsigned int posicion_segundo = result - d_vec;

    double min_val = *result;

    hipFree(retorno_segundo_d);

    if (min_val == inf){
        return -1;
    }

    return posicion_segundo;
}

inline int buscar_siguiente (Complex complejo_primero, Complex complejo_segundo, Complex * puntos_d, float epsilon, int numElements, double * retorno_siguiente_d){
    // Lanzamos la ejecución del kernel

    buscar_siguiente_kernel <<<blocksPerGrid, threadsPerBlock>>> (complejo_primero, complejo_segundo, puntos_d, epsilon,
                                                                  retorno_siguiente_d, numElements);

    hipGetLastError();

    device_ptr<double> d_vec (retorno_siguiente_d);
    device_ptr<double> d_vec_final = d_vec + numElements;
    device_ptr<double> result = min_element(d_vec, d_vec_final);

    //Recuperamos la posición del valor mínimo.
    unsigned int posicion_siguiente = result - d_vec;

    double min_val = *result;

    if (min_val == inf){
        return -1;
    }

    return posicion_siguiente;
}

static __global__ void buscar_segundo_kernel(Complex primero, Complex * puntos, float epsilon,
                                             double * retorno, int n_puntos)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n_puntos){

        double dist = abs(primero - puntos[i]);

        if (dist > 0 && dist <= epsilon){

            retorno[i] = arg(puntos[i] - primero) ;

            if (retorno[i] < 0){
                retorno[i] += 2 * M_PI;
            }

            retorno[i] -= acos(dist / epsilon);
        } else {
            retorno[i] = inf;
        }
    }
}

static __global__ void buscar_siguiente_kernel(Complex primero, Complex segundo, Complex * puntos, float epsilon,
                                               double * retorno, int n_puntos)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n_puntos){

        double dist = abs(puntos[i] - segundo);

        if (dist > 0 && dist <= epsilon && puntos[i] != primero && puntos[i] != segundo){

            double fase = arg((puntos[i] - segundo) / (primero - segundo));

            if (fase < 0){
                fase += 2 * M_PI;
            }

            double aco1 = acos(dist / epsilon);
            double aco2 = acos(abs(primero - segundo) / epsilon);

            if (fase == 0){
                retorno[i] = 2 * M_PI - aco1 - aco2;
            } else if (fase < aco2){
                retorno[i] = fase + aco1 - aco2;
            } else{
                retorno[i] = fase - aco1 - aco2;
            }

            if (retorno[i] < 0){
                retorno[i] += 2 * M_PI;
            }

        } else {
            retorno[i] = inf;
        }
    }
}

//Funciones para operar con números complejos.
static __device__ __host__ inline double abs(Complex a)
{
    //return hypot((float)a.x, (float) a.y);
    
    float t,x = a.x,y = a.y;
    x = abs(x);
    y = abs(y);
    t = min(x,y);
    x = max(x,y);
    t = t/x;
    
    return x*sqrt(1+t*t);
}

static __device__ __host__ inline Complex operator-(Complex a, Complex b)
{
    a.x = a.x - b.x;
    a.y = a.y - b.y;
    return a;
}

static __device__ __host__ inline double arg(Complex a)
{
    return atan2(a.y, a.x);
}

static __device__ __host__ inline Complex operator/(Complex a, Complex b)
{
    Complex c;
    double i = pow(b.x,2) + pow(b.y,2);
    c.x = (a.x * b.x + a.y * b.y)/i;
    c.y = (a.y * b.x - a.x * b.y)/i;
    return c;
}

static __device__ __host__ inline bool operator!=(Complex a, Complex b)
{
    if ((a.x != b.x) && (a.y != b.y))
        return true;
    
    return false;
}